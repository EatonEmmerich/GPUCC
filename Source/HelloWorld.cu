
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
void random_ints(int* a, int N)
{
	int i;
	for (i = 0; i < N; ++i)
		a[i] = rand();
}

__global__ void add(int * a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

void printc(int* c, int N) {
	for(int i = 0; i < N; i++){
		std::cout << c[i];
	}
}

int main(void) {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int N = 2048*2048;
	int NUM_THREADS = 512;
	int size = N*sizeof(int);
	
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_b,size);
	hipMalloc((void **) &d_c,size);
	
	a = (int *)malloc(size);
	random_ints(a,N);
	b = (int *)malloc(size);
	c = (int *)malloc(size);
	random_ints(b,N);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	add<<<N/NUM_THREADS,NUM_THREADS>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c, size, hipMemcpyDeviceToHost);
	printc(c, N);

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
