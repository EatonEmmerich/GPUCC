
#include <hip/hip_runtime.h>

__global__ void mypa(float in[4],float t,float t2){
//	float t2 = 2.0;
//	float t = 1.0 - (threadIdx.x + blockIdx.x*blockDim.x)c -c -arch=sm_20 cudacode.cu
	for(int x = 0; x < 6; x ++){
		in[0] = (in[0]+in[1]+in[2]-in[3])*t;
		in[1] = (in[0]+in[1]-in[2]+in[3])*t;
		in[2] = (in[0]-in[1]+in[2]+in[3])*t;
		in[3] = (-in[0]+in[1]+in[2]-in[3])/t2;
	}
}

void wrap(float in[4],float t,float t2){
	mypa<<<1, 1>>>(in,t,t2);
}

void mycudaInit(float *in_d,float *in){
	hipMalloc((void **)&in_d,4*sizeof(float));
        hipMemcpy(in_d,in,4*sizeof(float),hipMemcpyHostToDevice);
}

void mycudaFree(float *in_d, float *in){
	hipMemcpy(in_d,in,4*sizeof(float),hipMemcpyDeviceToHost);
	hipFree(in_d);
}
